#include "hip/hip_runtime.h"
// STL
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm> 
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


// Misc.
#include <clara/clara.hpp>
//#include <fmt/format.h>




// Functions
std::vector<int> retrieveIntegers(const std::string &directory);
std::vector<int> retrieveIntegers(const std::string &directory,const char &delimiter);
std::vector<int> operation(std::vector<int> &array1, std::vector<int> &array2, const char &operation);
//void sum(const std::vector<int> &array1, const std::vector<int> &array2, std::vector<int> &result);
//void sub(const std::vector<int> &array1,const std::vector<int> &array2, std::vector<int> &result);
//void mul(const std::vector<int> &array1,const std::vector<int> &array2, std::vector<int> &result);
//void div(const std::vector<int> &array1,const std::vector<int> &array2, std::vector<int> &result);
void streamOut(const std::vector<int> &results, const std::string &outDir);

__global__ void sum(thrust::device_vector<int> &Dev_array1,thrust::device_vector<int> &Dev_array2, thrust::device_vector<int> &Dev_result, int n)
{
    int i=threadIdx.x;
    if (i < n)
        Dev_result.insert(Dev_result.begin()+i, Dev_array1[i]+ Dev_array2[i]);
    
}

__global__ void sub(thrust::device_vector<int> &Dev_array1,thrust::device_vector<int> &Dev_array2, thrust::device_vector<int> &Dev_result, int n)
{
    int i=threadIdx.x;
    if (i < n)
        Dev_result.insert(Dev_result.begin()+i, Dev_array1[i]- Dev_array2[i]);
    
}


__global__ void mul(thrust::device_vector<int> &Dev_array1,thrust::device_vector<int> &Dev_array2, thrust::device_vector<int> &Dev_result, int n )
{
    int i=threadIdx.x;
    
    if (i < n)
        Dev_result.insert(Dev_result.begin()+i,Dev_array1[i]* Dev_array2[i]);
    
}
__global__ void div (thrust::device_vector<int> &Dev_array1,thrust::device_vector<int> &Dev_array2, thrust::device_vector<int> &Dev_result, int n)
{
    int i=threadIdx.x;

    if (i < n)
        Dev_result.insert(Dev_result.begin()+i, Dev_array1[i]/Dev_array2[i]);
    
}



bool VALID_OPERATION = false;

int main (int argc, char **argv){
  
    std::vector<int> arry1,arry2,results;


    std::string dir1, dir2; 
    std::string outDir = "results.txt";
    char delim1 = ' ';
    char delim2 = ' ';
    char oper;
    auto parser = clara::Arg(dir1, "dir1")("The path of the first file") |
                  clara::Arg(oper,"oper")("The operator") |
                  clara::Arg(dir2, "dir2")("The path to the second file") |
                  clara::Opt(delim1, "delimiter")["-f"]("Delimiter if exists")|
                  clara::Opt(delim2, "delimiter")["-s"]("Delimiter if exists") |
                  clara::Opt(outDir, "output directory")["-e"]("Output Directory");


    auto result = parser.parse(clara::Args(argc, argv));
      if (!result){
         std::cerr << "Error in command line: " << result.errorMessage() << std::endl;
         return 1;
      }
      else {
        
        arry1 = retrieveIntegers(dir1,delim1);
        arry2 = retrieveIntegers(dir2,delim2);
        results = operation(arry1,arry2,oper);
       
        
    
        if(VALID_OPERATION) {
            streamOut(results,outDir);

            }
        }
         return 0;
      }


// This function takes a directory and a delimiter, parses through the txt file, and returns a vector containing the integers

std::vector<int> retrieveIntegers(const std::string &directory, const char &delimiter){
    std::ifstream file;
    std::vector<std::string> arry;
    std::vector<int> arryint;
    std::string temp;
    file.open(directory);
    if (file.is_open()){

         while (getline(file, temp ,delimiter)){
            arry.push_back(temp);
         }

        file.close();

    }

    else {
        std::cerr << "Unable to open file " << directory << "\n";
    }
    for (int i=0; i<arry.size();i++){
        arryint.push_back(std::stoi(arry[i]));
    }
    return arryint;
}

// Calculation function that takes the operator and perform the operation on both files
// In case of unequal lengths of arrays, the shorter array is concatenated with 0s in cases of subtraction and addition
// or 1s in cases of multiplication or division

std::vector<int> operation(std::vector<int> &array1, std::vector<int> &array2, const char &operation){

    std::vector<int> result;
    thrust::device_vector<int> Dev_result(result);
    thrust::device_vector<int> Dev_array1(array1) ;
    thrust::device_vector<int> Dev_array2(array2);

    

    switch (operation){
        case '+':   VALID_OPERATION = true;
                    if (array1.size() > array2.size()){
                        for (int i=0; i < (array1.size() - array2.size()); i++){
                            array2.push_back(0);
                        } 
                    }  
                    else if ( array2.size() > array1.size() ) {
                        for (int i=0; i < (array2.size() - array1.size()); i++){
                            array1.push_back(0);
                        }    
                    }
                    sum<<<1,array1.size()>>>(Dev_array1,Dev_array2,Dev_result, array1.size());
                    return result;

        case '-':   VALID_OPERATION = true;
                    if (array1.size() > array2.size()){
                        for (int i=0; i < (array1.size() - array2.size()); i++){
                            array2.push_back(0);
                        } 
                    }  
                    else if ( array2.size() > array1.size() ) {
                        for (int i=0; i < (array2.size() - array1.size()); i++){
                            array1.push_back(0);
                        }    
                    }
                    sub<<<1,array1.size()>>>(Dev_array1,Dev_array2,Dev_result,array1.size());
                    return result;

        case '*':   VALID_OPERATION = true;
                    if (array1.size() > array2.size()){
                        for (int i=0; i < (array1.size() - array2.size()); i++){
                            array2.push_back(1);
                        } 
                    }  
                    else if ( array2.size() > array1.size() ) {
                        for (int i=0; i < (array2.size() - array1.size()); i++){
                            array1.push_back(1);
                        }    
                    }
                    mul<<<1,array1.size()>>>(Dev_array1,Dev_array2,Dev_result, array1.size());
                    return result;

        case '/':   VALID_OPERATION = true;
                    if (array1.size() > array2.size()){
                        for (int i=0; i < (array1.size() - array2.size()); i++){
                            array2.push_back(1);
                        } 
                    }  
                    else if ( array2.size() > array1.size() ) {
                        for (int i=0; i < (array2.size() - array1.size()); i++){
                            array1.push_back(1);
                        }    
                    }
                    div<<<1,array1.size()>>>(Dev_array1,Dev_array2,Dev_result,array1.size());
                    return result;
        
        default:    std::cout << "Invalid operator!" << "\n" << "Expected '+', '-', '*' or '/' " << "got " << operation << std::endl;
                    std::vector<int> emptyVector;
                    return emptyVector;
    }
}





// The function that export the results into a .txt file called "results.txt"

void streamOut(const std::vector<int> &results, const std::string &outDir){
    std::ofstream resultFile;
    resultFile.open(outDir);
    if (resultFile.is_open()){
        for(int i=0; i < results.size(); i++){
            resultFile << results[i] << " ";
        }
        resultFile.close();
    } else {
        std::cout << "Unable to open file";
    }
}



