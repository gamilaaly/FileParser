#include "hip/hip_runtime.h"
// STL
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <stdio.h>
#include <bits/stdc++.h>

#define SIZE 1024
using namespace std;
// Misc.
#include <clara/clara.hpp>
//#include <fmt/format.h>

//functions
int *retrieveIntegers(const std::string &directory, const char &delimiter);
int *operation(int *array1, int *array2, const char &operation);
void streamOut(int *results, const std::string &outDir);

__global__ void sum(int *array1, int *array2, int *results)
{   

    int i=threadIdx.x;
    if (i <SIZE)    {
        results[i] = array1[i] + array2[i];

    
    }
}
__global__ void sub(int *array1, int *array2, int *results)
{
    int i=threadIdx.x;
    if (i <SIZE)
    {
        results[i] = array1[i] - array2[i];
    }
}
__global__ void mul(int *array1, int *array2, int *results)
{
    int i=threadIdx.x;
    if (i <SIZE){
        results[i] = array1[i] * array2[i];
    }
}
__global__ void div(int *array1, int *array2, int *results)
{
    int i=threadIdx.x;
    if (i <SIZE) {
        results[i] = array1[i] / array2[i];
    }
}

bool VALID_OPERATION = false;

int main(int argc, char **argv)
{

    std::string dir1, dir2;
    std::string outDir = "results.txt";
    char delim1 = ' ';
    char delim2 = ' ';
    char oper;
    auto parser = clara::Arg(dir1, "dir1")("The path of the first file") |
                  clara::Arg(oper, "oper")("The operator") |
                  clara::Arg(dir2, "dir2")("The path to the second file") |
                  clara::Opt(delim1, "delimiter")["-f"]("Delimiter if exists") |
                  clara::Opt(delim2, "delimiter")["-s"]("Delimiter if exists") |
                  clara::Opt(outDir, "output directory")["-e"]("Output Directory");

    auto result = parser.parse(clara::Args(argc, argv));
    if (!result)
    {
        std::cerr << "Error in command line: " << result.errorMessage() << std::endl;
        return 1;
    }
    else
    {
        int *arry1, *arry2, *results;
        hipMallocManaged(&arry1 , SIZE*sizeof(int));
        hipMallocManaged(&arry2 , SIZE*sizeof(int));
        hipMallocManaged(&results, SIZE*sizeof(int));

        arry1 = retrieveIntegers(dir1, delim1);
        arry2 = retrieveIntegers(dir2, delim2);
        results = operation(&arry1[0], &arry2[0], oper);
        
        if (VALID_OPERATION)
        {
            streamOut(&results[0], outDir);
        }
        hipFree(arry1);
        hipFree(arry2);
        hipFree(results);
    }

    return 0;
}

int *retrieveIntegers(const std::string &directory, const char &delimiter)
{
    std::ifstream file;
    std::string temp;

    std::vector<std::string> arry;
    int *arryint;
    string *a;
    hipMallocManaged(&arryint , SIZE*sizeof(int));
    hipMallocManaged(&a , SIZE*sizeof(string));
   
    file.open(directory);
    if (file.is_open())
    {
        while (getline(file, temp, delimiter))
        {
            arry.push_back(temp);
        }
        std::copy(arry.begin(), arry.end(), a);
        file.close();
    }

    else
    {
        std::cerr << "Unable to open file " << directory << "\n";
    }

    for (int i = 0; i <10 ; i++)
    {

        arryint[i] = (std::stoi(a[i]));
    
    }
    return arryint;
}

int *operation(int *array1, int *array2, const char &operation)
{
    int *result ;
    hipMallocManaged(&result , SIZE*sizeof(int));


    switch (operation)
    {
    case 'p':
        VALID_OPERATION = true;
        {
            sum<<<1,SIZE>>>(&array1[0], &array2[0], &result[0]);
            hipDeviceSynchronize();
            return result;
        }

    case 's':
        VALID_OPERATION = true;
        {
            sub<<<1,SIZE>>>(&array1[0], &array2[0], &result[0]);
            hipDeviceSynchronize();
            return result;
        }

    case 'm':
        VALID_OPERATION = true;
        {
            mul<<<1,SIZE>>>(&array1[0], &array2[0], &result[0]);
            hipDeviceSynchronize();
            return result;
        }

    case 'd':
        VALID_OPERATION = true;
        {
            div<<<1,SIZE>>>(&array1[0], &array2[0], &result[0]);
            hipDeviceSynchronize();
            return result;
            
        }
        hipFree(result);

    default:
        std::cout << "Invalid operator!"
                  << "\n"
                  << "Expected '+', '-', '*' or '/' "
                  << "got " << operation << std::endl;
        int *emptyArray;

        return emptyArray;
    }

}

// The function that export the results into a .txt file called "results.txt"

void streamOut(int *results, const std::string &outDir)
{
    std::ofstream resultFile;
    resultFile.open(outDir);
    if (resultFile.is_open())
    {
        for (int i = 0; i < SIZE; i++)
        {
            resultFile << results[i] << " ";
        }
        resultFile.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }
}
