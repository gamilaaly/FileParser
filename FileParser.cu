#include "hip/hip_runtime.h"

/**
 * Asem: 
 * 1- The above paragraph is difficult to read. No need to upper case'ing it. (done)
 * */


/**
 * 
 * 2- The includes can be better sorted and made in related groups, I fixed this. done by Eng.Asem
 * 3- Be strictly consistent in using spaces, for example: done by Eng.Asem
 * Do not:
 * #include<fstream>
 * #include <vector>
 * Instead:
 * #include <fstream>
 * #include <vector>
 * I fixed this.
 * */

// STL
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm> 

// Misc.
#include <clara/clara.hpp>
#include <fmt/format.h>


/**
 * 4- Avoid opening namespaces global-wide.
 * You can instead open a namespace only within the function body, where you actually invoke the library. (done)
 * */



// Functions

/**
 * 11- A general rule for all of the functions in this file.
 * If you are receiving a container that you only read from, then receive it by constant reference (e.g const std::string &dictionary).
 * If you are receiving a container that you will modify, then receive it by reference (e.g std::string &dictionary).
 * By using references, you avoid the overhead of the unnecessary copying of data to a new container.
 * */

std::vector<int> retrieveIntegers(const std::string &directory);
std::vector<int> retrieveIntegers(const std::string &directory,const char &delimiter);
std::vector<int> operation(std::vector<int> &array1, std::vector<int> &array2, const char &operation);
void sum(const std::vector<int> &array1, const std::vector<int> &array2, std::vector<int> &result);
void sub(const std::vector<int> &array1,const std::vector<int> &array2, std::vector<int> &result);
void mul(const std::vector<int> &array1,const std::vector<int> &array2, std::vector<int> &result);
void div(const std::vector<int> &array1,const std::vector<int> &array2, std::vector<int> &result);
void streamOut(const std::vector<int> &results, const std::string &outDir);


/**
 * 5- Never make global variables, unless for defining global constants. // I don't get what is the problem here as we use it in main function and other function.
 * */
bool VALID_OPERATION = false;

int main (int argc, char **argv){
    std::vector<int> arry1,arry2,results;
    std::string dir1, dir2; // 6- Just a matter of taste, and to be consistent with the style at line 74.
    std::string outDir = "results.txt"; // 7- Always better to declare and initialize at the same line, when possible. 
    char delim1 = ' ';
    char delim2 = ' ';
    char oper;
    auto parser = clara::Arg(dir1, "dir1")("The path of the first file") |
                  clara::Arg(oper,"oper")("The operator") |
                  clara::Arg(dir2, "dir2")("The path to the second file") |
                  clara::Opt(delim1, "delimiter")["-f"]("Delimiter if exists")|
                  clara::Opt(delim2, "delimiter")["-s"]("Delimiter if exists") |
                  clara::Opt(outDir, "output directory")["-e"]("Output Directory");


    auto result = parser.parse(clara::Args(argc, argv));
      if (!result){
         std::cerr << "Error in command line: " << result.errorMessage() << std::endl;
         return 1;
      }
      else {
        
        arry1 = retrieveIntegers(dir1,delim1);
        arry2 = retrieveIntegers(dir2,delim2);
        results = operation(arry1,arry2,oper);


        if(VALID_OPERATION) {
            streamOut(results,outDir);

            }
        }
         return 0;
      }


// This function takes a directory and a delimiter, parses through the txt file, and returns a vector containing the integers

std::vector<int> retrieveIntegers(const std::string &directory, const char &delimiter){
    std::ifstream file;
    std::vector<std::string> arry;
    std::vector<int> arryint;
    std::string temp;
    file.open(directory);
    if (file.is_open()){

         while (getline(file, temp ,delimiter)){
            arry.push_back(temp);
         }

        file.close();

    }

    else {
        std::cerr << "Unable to open file " << directory << "\n";
    }
    for (int i=0; i<arry.size();i++){
        arryint.push_back(std::stoi(arry[i]));
    }
    return arryint;
}

// Calculation function that takes the operator and perform the operation on both files
// In case of unequal lengths of arrays, the shorter array is concatenated with 0s in cases of subtraction and addition
// or 1s in cases of multiplication or division

std::vector<int> operation(std::vector<int> &array1, std::vector<int> &array2, const char &operation){

    std::vector<int> result;
    /**
     * 9- Here you better represent the operation with `char` type (e.g '+', '-', etc.) to 
     * avoid the misspelling of words, or ideally, in big projects, using Enum Classes.
     * By using `char` or Enums, here we optimally can use the switch-case statement that 
     * provides a more readable logic. (done)
     * */
    switch (operation){
        case '+':   VALID_OPERATION = true;
                    if (array1.size() > array2.size()){
                        for (int i=0; i < (array1.size() - array2.size()); i++){
                            array2.push_back(0);
                        } 
                    }  
                    else if ( array2.size() > array1.size() ) {
                        for (int i=0; i < (array2.size() - array1.size()); i++){
                            array1.push_back(0);
                        }    
                    }
                    sum<<<1,1>>>(array1,array2,result);
                    return result;

        case '-':   VALID_OPERATION = true;
                    if (array1.size() > array2.size()){
                        for (int i=0; i < (array1.size() - array2.size()); i++){
                            array2.push_back(0);
                        } 
                    }  
                    else if ( array2.size() > array1.size() ) {
                        for (int i=0; i < (array2.size() - array1.size()); i++){
                            array1.push_back(0);
                        }    
                    }
                    sub<<<1,1>>>(array1,array2,result);
                    return result;

        case '*':   VALID_OPERATION = true;
                    if (array1.size() > array2.size()){
                        for (int i=0; i < (array1.size() - array2.size()); i++){
                            array2.push_back(1);
                        } 
                    }  
                    else if ( array2.size() > array1.size() ) {
                        for (int i=0; i < (array2.size() - array1.size()); i++){
                            array1.push_back(1);
                        }    
                    }
                    mul<<<1,1>>>(array1,array2,result);
                    return result;

        case '/':   VALID_OPERATION = true;
                    if (array1.size() > array2.size()){
                        for (int i=0; i < (array1.size() - array2.size()); i++){
                            array2.push_back(1);
                        } 
                    }  
                    else if ( array2.size() > array1.size() ) {
                        for (int i=0; i < (array2.size() - array1.size()); i++){
                            array1.push_back(1);
                        }    
                    }
                    div<<<1,1>>>(array1,array2,result);
                    return result;
        
        default:    std::cout << "Invalid operator!" << "\n" << "Expected '+', '-', '*' or '/' " << "got " << operation << std::endl;
                    std::vector<int> emptyVector;
                    return emptyVector;
    }
}


__global__
void sum(const std::vector<int> &array1,const std::vector<int> &array2, std::vector<int> &result){
    for(int i=0; i<array1.size(); i++){
        result.push_back(array1[i]+array2[i]);
    }
}

__global__
void sub(const std::vector<int> &array1,const std::vector<int> &array2, std::vector<int> &result){
    for(int i=0; i<array1.size(); i++){
        result.push_back(array1[i]-array2[i]);
    }
}


__global__
void mul(const std::vector<int> &array1,const std::vector<int> &array2, std::vector<int> &result){
    for(int i=0; i<array1.size(); i++){
        result.push_back(array1[i]*array2[i]);
    }
}
__global__
void div(const std::vector<int> &array1,const std::vector<int> &array2, std::vector<int> &result){
    for(int i=0; i<array1.size(); i++){
        result.push_back(array1[i]/array2[i]);
    }
}


// The function that export the results into a .txt file called "results.txt"

void streamOut(const std::vector<int> &results, const std::string &outDir){
    std::ofstream resultFile;
    resultFile.open(outDir);
    if (resultFile.is_open()){
        for(int i=0; i < results.size(); i++){
            resultFile << results[i] << " ";
        }
        resultFile.close();
    } else {
        std::cout << "Unable to open file";
    }
}



